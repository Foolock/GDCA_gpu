#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>

void readDumpFile(const std::string& filename, std::vector<int>& vector1, std::vector<int>& vector2) {

  std::ifstream file(filename);
  if (!file.is_open()) {
    std::cerr << "Failed to open the file.\n";
    return;
  }

  std::string line;

  // Read the first line of integers
  if (std::getline(file, line)) {
    std::istringstream stream(line);
    int num;
    while (stream >> num) {
      vector1.push_back(num);
    }
  }

  // Read the second line of integers
  if (std::getline(file, line)) {
    std::istringstream stream(line);
    int num;
    while (stream >> num) {
      vector2.push_back(num);
    }
  }

  file.close();
}

__global__ void csr_graph(const int* adjp_gpu, const int* adjncy_gpu, int N, int M) {

  /*
  // check csr
  printf("adjp_gpu = [");
  for(size_t i=0; i<N; i++) {
    printf("%d ", adjp_gpu[i]);
  }
  printf("]\n");
  printf("adjncy_gpu = [");
  for(size_t i=0; i<M; i++) {
    printf("%d ", adjncy_gpu[i]);
  }
  printf("]\n");
  */

}

int main() {

  /*
   * read graph information
   */

  std::vector<int> adjp, adjncy;
  std::string csr_path = "../../csr_data/csr.dmp"; // Change this to your file path
  readDumpFile(csr_path, adjp, adjncy); 

  /*
  // check csr
  std::cerr << "adjp = [";
  for(auto id : adjp) {
    std::cerr << id << " ";
  }
  std::cerr << "]\n";
  std::cerr << "adjncy = [";
  for(auto id : adjncy) {
    std::cerr << id << " ";
  }
  std::cerr << "]\n";
  */

  /*
   * transfer csr data to gpu
   */

  int* adjp_gpu; 
  int* adjncy_gpu; 
  hipMalloc(&adjp_gpu, sizeof(int)*adjp.size());
  hipMalloc(&adjncy_gpu, sizeof(int)*adjncy.size());
  hipMemcpy(adjp_gpu, adjp.data(), sizeof(int)*adjp.size(), hipMemcpyDefault);
  hipMemcpy(adjncy_gpu, adjncy.data(), sizeof(int)*adjncy.size(), hipMemcpyDefault);

  unsigned num_block = 1; 	
  unsigned num_threads = 1;
 
  csr_graph<<<num_block, num_threads>>>(adjp_gpu, adjncy_gpu, adjp.size(), adjncy.size()); 

  hipDeviceSynchronize();

  return 0;
}

