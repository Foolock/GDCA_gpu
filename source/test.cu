#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>

void readDumpFile(const std::string& filename, std::vector<int>& vector1, std::vector<int>& vector2) {

  std::ifstream file(filename);
  if (!file.is_open()) {
    std::cerr << "Failed to open the file.\n";
    return;
  }

  std::string line;

  // Read the first line of integers
  if (std::getline(file, line)) {
    std::istringstream stream(line);
    int num;
    while (stream >> num) {
      vector1.push_back(num);
    }
  }

  // Read the second line of integers
  if (std::getline(file, line)) {
    std::istringstream stream(line);
    int num;
    while (stream >> num) {
      vector2.push_back(num);
    }
  }

  file.close();
}
__global__ void kernel1() {

  int num = threadIdx.x + 1; // a number between 1 to 8  
  int result = 1; // factorial result of the number

  for(int i=1; i<=num; i++) {
    result = result * i;
  }
  std::printf("%d!=%d\n", num, result);
   
}

int main() {

  /*
   * read graph information
   */

  std::vector<int> _adjp, _adjncy;
  std::string csr_path = "../../csr_data/csr.dmp"; // Change this to your file path

  readDumpFile(csr_path, _adjp, _adjncy); 

  // check csr
  std::cerr << "_adjp = [";
  for(auto id : _adjp) {
    std::cerr << id << " ";
  }
  std::cerr << "]\n";
  std::cerr << "_adjncy = [";
  for(auto id : _adjncy) {
    std::cerr << id << " ";
  }
  std::cerr << "]\n";

  unsigned num_block = 1; 	
  unsigned num_threads = 8;

  kernel1<<<num_block, num_threads>>>();

  hipDeviceSynchronize();

  return 0;
}

